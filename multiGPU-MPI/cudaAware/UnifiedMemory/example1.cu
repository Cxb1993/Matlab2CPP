
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printValue( int *value) {
printf("value %d\n",value[0]);
printf("value %d\n",value[1]);
}
 
void hostFunction(){
int *value;
hipMallocManaged(&value, 2 * sizeof(int));
value[0]=1;
value[1]=2;
printValue<<< 1, 1 >>>(value);
hipDeviceSynchronize();
hipFree(value);
}
 
int main() {
hostFunction();
return 0;
}
