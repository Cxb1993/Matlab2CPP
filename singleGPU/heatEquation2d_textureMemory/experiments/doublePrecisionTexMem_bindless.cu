// Take From 
// https://stackoverflow.com/questions/35137213/texture-objects-for-doubles


#include <hip/hip_runtime.h>
#include <vector>
#include <cstdio>

static __inline__ __device__ double fetch_double(uint2 p){
    return __hiloint2double(p.y, p.x);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void my_print(hipTextureObject_t texObject)
{
    uint2 rval = tex1Dfetch<uint2>(texObject, 0);
    double dval = fetch_double(rval);
    printf("%f\n", dval);
}

int main()
{

    double i = 0.35;
    int numel = 50;

    std::vector<double> h_data(numel, i);
    double* d_data;
    hipMalloc(&d_data,numel*sizeof(double));
    hipMemcpy((void*)d_data, &h_data[0], numel*sizeof(double), hipMemcpyHostToDevice);


    hipTextureDesc td;
    memset(&td, 0, sizeof(td));
    td.normalizedCoords = 0;
    td.addressMode[0] = hipAddressModeClamp;
    td.readMode = hipReadModeElementType;


    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_data;
    resDesc.res.linear.sizeInBytes = numel*sizeof(double);
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32;
    resDesc.res.linear.desc.y = 32;

    hipTextureObject_t texObject;
    gpuErrchk(hipCreateTextureObject(&texObject, &resDesc, &td, NULL));

    my_print<<<1,1>>>(texObject);

    gpuErrchk(hipDeviceSynchronize());
    return 0;
}