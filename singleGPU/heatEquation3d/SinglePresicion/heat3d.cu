#include "hip/hip_runtime.h"

#include "heat3d.h"

/***********************/
/* AUXILIARY FUCNTIONS */
/***********************/
void Print2D(float *u)
{
    // print a single property on terminal
    for (int k = 0; k < NZ; k++) {
      for (int j = 0; j < NY; j++) {
        for (int i = 0; i < NX; i++) {
          printf("%8.2f", u[i+NX*j+NX*NY*k]);
        }
        printf("\n");
      }
      printf("\n\n");
    }
}

void Save_Results(float *u){
  // print result to txt file
  float data;
  FILE *pFile = fopen("result.txt", "w");  
  int XY=NX*NY;
  if (pFile != NULL) {
    for (int k = 0; k < NZ; k++) {
      for (int j = 0; j < NY; j++) {
        for (int i = 0; i < NX; i++) {
          data = u[i+NX*j+XY*k];
          fprintf(pFile, "%d\t %d\t %d\t %g\n",k,j,i,data);
        }
      }
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/

void Call_IC(float *__restrict u0){
  int i, j, k, o, IC, XY=NX*NY;

  // select IC
  IC=2;

  switch (IC) {
  case 1: {
    for (k = 0; k < NZ; k++) {
      for (j = 0; j < NY; j++) {
      	for (i = 0; i < NX; i++) {
      	  // set all domain's cells equal to zero
      	  o = i+NX*j+XY*k;  u0[o] = 0.0;
      	  // set BCs in the domain 
      	  if (k==0)    u0[o] = 1.0; // bottom
      	  if (k==NZ-1) u0[o] = 1.0; // top
      	}
      }
    }
    break;
  }
  case 2: {
    for (k = 0; k < NZ; k++) {
      for (j = 0; j < NY; j++) {
        for (i = 0; i < NX; i++) {
          // set all domain's cells equal to zero
          o = i+NX*j+XY*k;  
          u0[o] = 1.0*exp(
            -(DX*(i-NX/2))*(DX*(i-NX/2))/1.5
            -(DY*(j-NY/2))*(DY*(j-NY/2))/1.5
            -(DZ*(k-NZ/2))*(DZ*(k-NZ/2))/12);
        }
      }
    }
    break;
  }
    // here to add another IC
  } 
}

/************************************/
/* LAPLACE ITERATION FUNCTION - CPU */
/************************************/

void Laplace3d_CPU(float *u, float *un){
  // Using (i,j) = [i+N*j+M*N*k] indexes
  int i, j, k, o, n, s, e, w, t, b; 
  int XY=NX*NY;

  for (j = 0; j < NY; j++) {
    for (i = 0; i < NX; i++) {
      for (k = 0; k < NZ; k++) {
	
        o = i+ (NX*j) + (XY*k); // node( j,i,k )      n  b
        n = (i==NX-1) ? o:o+NX; // node(j+1,i,k)      | /
        s = (i==0)    ? o:o-NX; // node(j-1,i,k)      |/
        e = (j==NY-1) ? o:o+1;  // node(j,i+1,k)  w---o---e
        w = (j==0)    ? o:o-1;  // node(j,i-1,k)     /|
        t = (k==NZ-1) ? o:o+XY; // node(j,i,k+1)    / |
        b = (k==0)    ? o:o-XY; // node(j,i,k-1)   t  s

        un[o] = u[o] + KX*(u[e]-2*u[o]+u[w]) + KY*(u[n]-2*u[o]+u[s]) + KZ*(u[t]-2*u[o]+u[b]);
      }
    } 
  }
}

/************************************/
/* LAPLACE ITERATION FUNCTION - GPU */
/************************************/

__global__ void Laplace3d_GPU(const float * __restrict__ u, float * __restrict__ un){
  int o, n, s, e, w, t, b;  
  int XY=NX*NY;
  // Threads id
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  o = i+ (NX*j) + (XY*k); // node( j,i,k )      n  b
  n = (i==NX-1) ? o:o+NX; // node(j+1,i,k)      | /
  s = (i==0)    ? o:o-NX; // node(j-1,i,k)      |/
  e = (j==NY-1) ? o:o+1;  // node(j,i+1,k)  w---o---e
  w = (j==0)    ? o:o-1;  // node(j,i-1,k)     /|
  t = (k==NZ-1) ? o:o+XY; // node(j,i,k+1)    / |
  b = (k==0)    ? o:o-XY; // node(j,i,k-1)   t  s

  un[o] = u[o] + KX*(u[e]-2*u[o]+u[w]) + KY*(u[n]-2*u[o]+u[s]) + KZ*(u[t]-2*u[o]+u[b]);
}

void Call_Laplace(dim3 numBlocks, dim3 threadsPerBlock, float *d_u, float *d_un) {
  // Produce one iteration of the laplace operator

  Laplace3d_GPU<<<numBlocks,threadsPerBlock>>>(d_u,d_un);
  Laplace3d_GPU<<<numBlocks,threadsPerBlock>>>(d_un,d_u);
  if (DEBUG) printf("CUDA error (Laplace GPU %d) %s\n",
    hipGetErrorString(hipPeekAtLastError()));

  hipError_t Error = hipDeviceSynchronize();
  if (DEBUG) printf("CUDA error (Laplace GPU %d Synchronize) %s\n",
    hipGetErrorString(Error));
}
