#include "hip/hip_runtime.h"

#include "heat2d.h"

void Manage_Memory(int phase, int tid, float **h_u, float **d_u, float **d_un){
  if (phase==0) {
    // Allocate whole domain in host (master thread)
    *h_u = (float*)malloc(NY*NX*sizeof(float));
  }
  if (phase==1) {
    // Allocate whole domain in device (GPU thread)
    hipError_t Error = hipSetDevice(tid);
    if (DEBUG) printf("CUDA error (hipSetDevice) = %s\n",hipGetErrorString(Error));
    Error = hipMalloc((void**)d_u ,NY*NX*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc) = %s\n",hipGetErrorString(Error));
    Error = hipMalloc((void**)d_un,NY*NX*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc) = %s\n",hipGetErrorString(Error));
  }
  if (phase==2) {
    // Free the whole domain variables (master thread)
    free(*h_u);
    hipError_t Error;
    Error = hipFree(*d_u);
    if (DEBUG) printf("CUDA error (hipFree) = %s\n",hipGetErrorString(Error));
    Error = hipFree(*d_un);
    if (DEBUG) printf("CUDA error (hipFree) = %s\n",hipGetErrorString(Error));
  }
}

void Manage_Comms(int phase, int tid, float **h_u, float **d_u) {
  // Manage CPU-GPU communicastions
  if (DEBUG) printf(":::::::: Performing Comms (phase %d) ::::::::\n",phase);
  
  if (phase == 0) {
    // move h_u (from HOST) to d_u (to GPU)
    hipError_t Error = hipMemcpy(*d_u,*h_u,NY*NX*sizeof(float),hipMemcpyHostToDevice);
    if (DEBUG) printf("CUDA error (memcpy h -> d ) = %s\n",hipGetErrorString(Error));
  }
  if (phase == 1) {
    // move d_u (from GPU) to h_u (to HOST)
    hipError_t Error = hipMemcpy(*h_u,*d_u,NY*NX*sizeof(float),hipMemcpyDeviceToHost);
    if (DEBUG) printf("CUDA error (memcpy d -> h ) = %s\n",hipGetErrorString(Error));
  }
}

void Save_Results(float *u){
  // print result to txt file
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int j = 0; j < NY; j++) {
      for (int i = 0; i < NX; i++) {      
	fprintf(pFile, "%d\t %d\t %g\n",j,i,u[i+NX*j]);
      }
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}

__global__ void SetIC_onDevice(float *u){
  // threads id 
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int o = i+NX*j; u[o] = 0.0;
  // but ...
  if (i==0)    u[o] = 0.0;
  if (j==0)    u[o] = 0.0;
  if (i==NX-1) u[o] = 1.0;
  if (j==NY-1) u[o] = 1.0;
}

void Call_GPU_Init(float **u0){
  // Load the initial condition
  dim3 threads(16,16);
  dim3 blocks((NX+16+1)/16,(NY+16+1)/16); 
  SetIC_onDevice<<<blocks, threads>>>(*u0);
}

__global__ void Laplace2d(const float * __restrict__ u, float * __restrict__ un){
  int o, n, s, e, w; 
  // Threads id
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  const int j = threadIdx.y + blockIdx.y*blockDim.y;

  o = i + (NX*j);         // node( j,i,k )      n
  n = (i==NX-1) ? o:o+NX; // node(j+1,i,k)      |
  s = (i==0)    ? o:o-NX; // node(j-1,i,k)   w--o--e
  e = (j==NY-1) ? o:o+1;  // node(j,i+1,k)      |
  w = (j==0)    ? o:o-1;  // node(j,i-1,k)      s

  // only update "interior" nodes
  if(i>0 && i<NX-1 && j>0 && j<NY-1) {
    un[o] = u[o] + KX*(u[e]-2*u[o]+u[w]) + KY*(u[n]-2*u[o]+u[s]);
  } else {
    un[o] = u[o];
  }
}

void Call_Laplace(float **d_u, float **d_un) {
  // Produce one iteration of the laplace operator
  dim3 threads(16,16);
  dim3 blocks((NX+16+1)/16,(NX+16+1)/16); 
  Laplace2d<<<blocks,threads>>>(*d_u,*d_un);
  if (DEBUG) printf("CUDA error (Jacobi_Method) %s\n",hipGetErrorString(hipPeekAtLastError()));
  hipError_t Error = hipDeviceSynchronize();
  if (DEBUG) printf("CUDA error (Jacobi_Method Synchronize) %s\n",hipGetErrorString(Error));
}
