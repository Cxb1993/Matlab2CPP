#include "hip/hip_runtime.h"

#include "heat2d.h"

void Manage_Memory(int phase, int tid, float **h_u, float **d_u, float **d_un){
  if (phase==0) {
    // Allocate whole domain in host (master thread)
    *h_u = (float*)malloc(NY*NX*sizeof(float));
  }
  if (phase==1) {
    // Allocate whole domain in device (GPU thread)
    hipError_t Error = hipSetDevice(tid);
    if (DEBUG) printf("CUDA error (hipSetDevice) = %s\n",hipGetErrorString(Error));
    Error = hipMalloc((void**)d_u ,NY*NX*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc) = %s\n",hipGetErrorString(Error));
    Error = hipMalloc((void**)d_un,NY*NX*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc) = %s\n",hipGetErrorString(Error));
  }
  if (phase==2) {
    // Free the whole domain variables (master thread)
    free(*h_u);
    hipError_t Error;
    Error = hipFree(*d_u);
    if (DEBUG) printf("CUDA error (hipFree) = %s\n",hipGetErrorString(Error));
    Error = hipFree(*d_un);
    if (DEBUG) printf("CUDA error (hipFree) = %s\n",hipGetErrorString(Error));
  }
}

void Manage_Comms(int phase, int tid, float **h_u, float **d_u) {
  // Manage CPU-GPU communicastions
  if (DEBUG) printf(":::::::: Performing Comms (phase %d) ::::::::\n",phase);
  
  if (phase == 0) {
    // move h_u (from HOST) to d_u (to GPU)
    hipError_t Error = hipMemcpy(*d_u,*h_u,NY*NX*sizeof(float),hipMemcpyHostToDevice);
    if (DEBUG) printf("CUDA error (memcpy h -> d ) = %s\n",hipGetErrorString(Error));
  }
  if (phase == 1) {
    // move d_u (from GPU) to h_u (to HOST)
    hipError_t Error = hipMemcpy(*h_u,*d_u,NY*NX*sizeof(float),hipMemcpyDeviceToHost);
    if (DEBUG) printf("CUDA error (memcpy d -> h ) = %s\n",hipGetErrorString(Error));
  }
}

void Save_Results(float *u){
  // print result to txt file
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int j = 0; j < NY; j++) {
      for (int i = 0; i < NX; i++) {      
	fprintf(pFile, "%d\t %d\t %g\n",j,i,u[i+NX*j]);
      }
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
__global__ void SetIC_onDevice(float *u0){
int i, j, o, IC; 
  // threads id 
  i = threadIdx.x + blockIdx.x*blockDim.x;
  j = threadIdx.y + blockIdx.y*blockDim.y;

  // select IC
  IC=2;

  switch (IC) {
  case 1: {
	// set all domain's cells equal to zero
	o = i+NX*j;  u0[o] = 0.0;
	// set BCs in the domain 
	if (j==0)    u0[o] = 0.0; // bottom
	if (i==0)    u0[o] = 0.0; // left
	if (j==NY-1) u0[o] = 1.0; // top
	if (i==NX-1) u0[o] = 1.0; // right
    break;
  }
  case 2: {
    float u_bl = 0.7f;
    float u_br = 1.0f;
    float u_tl = 0.7f;
    float u_tr = 1.0f;

	// set all domain's cells equal to zero
	o = i+NX*j;  u0[o] = 0.0;
	// set BCs in the domain 
	if (j==0)    u0[o] = u_bl + (u_br-u_bl)*i/(NX+1); // bottom
	if (j==NY-1) u0[o] = u_tl + (u_tr-u_tl)*i/(NX+1); // top
	if (i==0)    u0[o] = u_bl + (u_tl-u_bl)*j/(NY+1); // left
	if (i==NX-1) u0[o] = u_br + (u_tr-u_br)*j/(NY+1); // right
    break;
  }
  case 3: {
	// set all domain's cells equal to zero
	o = i+NX*j;  u0[o] = 0.0;
	// set left wall to 1
	if (i==NX-1) u0[o] = 1.0;
    break;
  }
    // here to add another IC
  }
}

void Call_GPU_Init(float **u0){
  // Load the initial condition
  dim3 threads(32,32);
  dim3 blocks((NX+1)/32,(NY+1)/32); 
  SetIC_onDevice<<<blocks, threads>>>(*u0);
}

__global__ void Laplace2d(const float * __restrict__ u, float * __restrict__ un){
  int o, n, s, e, w; 
  // Threads id
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  const int j = threadIdx.y + blockIdx.y*blockDim.y;

  o = i + (NX*j);         // node( j,i,k )      n
  n = (i==NX-1) ? o:o+NX; // node(j+1,i,k)      |
  s = (i==0)    ? o:o-NX; // node(j-1,i,k)   w--o--e
  e = (j==NY-1) ? o:o+1;  // node(j,i+1,k)      |
  w = (j==0)    ? o:o-1;  // node(j,i-1,k)      s

  // only update "interior" nodes
  if(i>0 && i<NX-1 && j>0 && j<NY-1) {
    un[o] = u[o] + KX*(u[e]-2*u[o]+u[w]) + KY*(u[n]-2*u[o]+u[s]);
  } else {
    un[o] = u[o];
  }
}

void Call_Laplace(float **d_u, float **d_un) {
  // Produce one iteration of the laplace operator
  dim3 threads(32,32);
  dim3 blocks((NX+1)/32,(NY+1)/32); 
  Laplace2d<<<blocks,threads>>>(*d_u,*d_un);
  if (DEBUG) printf("CUDA error (Jacobi_Method) %s\n",hipGetErrorString(hipPeekAtLastError()));
  hipError_t Error = hipDeviceSynchronize();
  if (DEBUG) printf("CUDA error (Jacobi_Method Synchronize) %s\n",hipGetErrorString(Error));
}
