#include "hip/hip_runtime.h"

#include "heat2d.h"

/* Initialize textures */
// texture<float, 2, hipReadModeElementType> tex_u;
// texture<float, 2, hipReadModeElementType> tex_un;
texture<int2, 2, hipReadModeElementType> tex_u;
texture<int2, 2, hipReadModeElementType> tex_un;

static __inline__ __device__ double fetch_double(int2 p){
    return __hiloint2double(p.y, p.x);
}

/***********************/
/* AUXILIARY FUCNTIONS */
/***********************/
void Print2D(double *u, const unsigned int nx, const unsigned int ny)
{
    unsigned int i, j;
    // print a single property on terminal
    for (j = 0; j < ny; j++) {
        for (i = 0; i < nx; i++) {
            printf("%8.2f", u[i+nx*j]);
        }
        printf("\n");
    }
    printf("\n");
}

void Save_Results(double *u){
  // print result to txt file
  float data;
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int j = 0; j < NY; j++) {
      for (int i = 0; i < NX; i++) {      
        data = u[i+NX*j]; 
        fprintf(pFile, "%d\t %d\t %g\n",j,i,data);
      }
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
void Call_IC(double * __restrict u0){
  int i, j, o, IC; 

  // select IC
  IC=2;

  switch (IC) {
  case 1: {
    for (j = 0; j < NY; j++) {
      for (i = 0; i < NX; i++) {
        // set all domain's cells equal to zero
        o = i+NX*j;  u0[o] = 0.0;
        // set BCs in the domain 
        if (j==0)    u0[o] = 0.0; // bottom
        if (i==0)    u0[o] = 0.0; // left
        if (j==NY-1) u0[o] = 1.0; // top
        if (i==NX-1) u0[o] = 1.0; // right
      }
    }
    break;
  }
  case 2: {
    double u_bl = 0.7f;
    double u_br = 1.0f;
    double u_tl = 0.7f;
    double u_tr = 1.0f;

    for (j = 0; j < NY; j++) {
      for (i = 0; i < NX; i++) {
        // set all domain's cells equal to zero
        o = i+NX*j;  u0[o] = 0.0;
        // set BCs in the domain 
        if (j==0)    u0[o] = u_bl + (u_br-u_bl)*i/(NX+1); // bottom
        if (j==NY-1) u0[o] = u_tl + (u_tr-u_tl)*i/(NX+1); // top
        if (i==0)    u0[o] = u_bl + (u_tl-u_bl)*j/(NY+1); // left
        if (i==NX-1) u0[o] = u_br + (u_tr-u_br)*j/(NY+1); // right
      }
    }
    break;
  }
  case 3: {
    for (j = 0; j < NY; j++) {
      for (i = 0; i < NX; i++) {
  // set all domain's cells equal to zero
  o = i+NX*j;  u0[o] = 0.0;
  // set left wall to 1
  if (i==NX-1) u0[o] = 1.0;
      }
    }
    break;
  }
    // here to add another IC
  }
}

__global__ void Laplace2d_texture(double * __restrict__ un, const bool flag) {

  // Threads id
  const int i = blockIdx.x * blockDim.x + threadIdx.x ;
  const int j = blockIdx.y * blockDim.y + threadIdx.y ;  

  double o, n, s, e, w; 
  int2 uData;
  if (flag) {
    uData = tex2D(tex_u,i, j ); o = fetch_double(uData); // node( i,j )     n
    uData = tex2D(tex_u,i,j+1); n = fetch_double(uData); // node(i,j+1)     |
    uData = tex2D(tex_u,i,j-1); s = fetch_double(uData); // node(i,j-1)  w--o--e
    uData = tex2D(tex_u,i+1,j); e = fetch_double(uData); // node(i+1,j)     |
    uData = tex2D(tex_u,i-1,j); w = fetch_double(uData); // node(i-1,j)     s
  } else {
    uData = tex2D(tex_un,i, j ); o = fetch_double(uData); // node( i,j )     n
    uData = tex2D(tex_un,i,j+1); n = fetch_double(uData); // node(i,j+1)     |
    uData = tex2D(tex_un,i,j-1); s = fetch_double(uData); // node(i,j-1)  w--o--e
    uData = tex2D(tex_un,i+1,j); e = fetch_double(uData); // node(i+1,j)     |
    uData = tex2D(tex_un,i-1,j); w = fetch_double(uData); // node(i-1,j)     s
  }

  // float o, n, s, e, w;
  // if (flag) {
  //   o = tex2D(tex_u,i, j ); // node( i,j )     n
  //   n = tex2D(tex_u,i,j+1); // node(i,j+1)     |
  //   s = tex2D(tex_u,i,j-1); // node(i,j-1)  w--o--e
  //   e = tex2D(tex_u,i+1,j); // node(i+1,j)     |
  //   w = tex2D(tex_u,i-1,j); // node(i-1,j)     s
  // } else {
  //   o = tex2D(tex_un,i, j ); // node( i,j )     n
  //   n = tex2D(tex_un,i,j+1); // node(i,j+1)     |
  //   s = tex2D(tex_un,i,j-1); // node(i,j-1)  w--o--e
  //   e = tex2D(tex_un,i+1,j); // node(i+1,j)     |
  //   w = tex2D(tex_un,i-1,j); // node(i-1,j)     s
  // }
  // --- Only update "interior" (not boundary) node points
  if (i>0 && i<NX-1 && j>0 && j<NY-1) un[i+j*NX] = o + KX*(e-2*o+w) + KY*(n-2*o+s);
}

void Call_Laplace(dim3 numBlocks, dim3 threadsPerBlock, double *d_u, double *d_un) 
{
  // Configure and Bind Textures to global memory
  // hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  // hipChannelFormatDesc desc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipChannelFormatDesc desc = hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindSigned);

  tex_u.addressMode[0] = hipAddressModeWrap;
  tex_u.addressMode[1] = hipAddressModeWrap;
  tex_u.filterMode = hipFilterModePoint;
  tex_u.normalized = false;
  
  tex_un.addressMode[0] = hipAddressModeWrap;
  tex_un.addressMode[1] = hipAddressModeWrap;
  tex_un.filterMode = hipFilterModePoint;
  tex_un.normalized = false;

  // bind textures
  // hipBindTexture2D(0,&tex_u, d_u, &desc,NX,NY,NX*sizeof(float));
  // hipBindTexture2D(0,&tex_un,d_un,&desc,NX,NY,NX*sizeof(float));
  hipBindTexture2D(0,&tex_u, d_u, &desc,NX,NY,NX*sizeof(double));
  hipBindTexture2D(0,&tex_un,d_un,&desc,NX,NY,NX*sizeof(double));

  // Produce one iteration of the laplace operator
  Laplace2d_texture<<<numBlocks,threadsPerBlock>>>(d_u, 0);
  Laplace2d_texture<<<numBlocks,threadsPerBlock>>>(d_un,1);

  // Unbind textures
  hipUnbindTexture(tex_u);
  hipUnbindTexture(tex_un);
}
