#include "hip/hip_runtime.h"
#include "cudamatrix_types.cuh"
#include <hiprand/hiprand_kernel.h>
#include "hiprand.h"

texture<float,hipTextureType2DLayered,hipReadModeElementType> texref;

__global__
void fill_kernel(cudaMatrixf data,int3 dims,cudaMatrixT<hiprandState>random_states)
{

	unsigned int idx = threadIdx.x;
	unsigned int idy = threadIdx.y;
	unsigned int idz = threadIdx.z;

	unsigned int gidx = blockIdx.x*blockDim.x+idx;
	unsigned int gidy = blockIdx.y*blockDim.y+idy;
	unsigned int gidz  = blockIdx.z*blockDim.z+idz;

	if((gidx < dims.x)&&(gidy < dims.y)&&(gidz < dims.z))
	{
		data(gidx,gidy,gidz) = (hiprand_uniform(&random_states(gidx,gidy,gidz))*100);
	}

}

__global__
void setup_kernel(cudaMatrixT<hiprandState> random_states)
{

	unsigned int idx = threadIdx.x;
	unsigned int idy = threadIdx.y;
	unsigned int idz = threadIdx.z;

	unsigned int gidx = blockIdx.x*blockDim.x+idx;
	unsigned int gidy = blockIdx.y*blockDim.y+idy;
	unsigned int gidz  = blockIdx.z*blockDim.z+idz;

	hiprand_init(6446574,gidx+gridDim.x*(gidy+gridDim.y*gidz),0,&random_states(gidx,gidy,gidz));

}

__global__
void check_kernel(cudaMatrixf data,int3 dims)
{

	unsigned int idx = threadIdx.x;
	unsigned int idy = threadIdx.y;
	unsigned int idz = threadIdx.z;

	unsigned int gidx = blockIdx.x*blockDim.x+idx;
	unsigned int gidy = blockIdx.y*blockDim.y+idy;
	unsigned int gidz  = blockIdx.z*blockDim.z+idz;

	float mydata;
	float texdata;

	if((gidx < dims.x)&&(gidy < dims.y)&&(gidz < dims.z))
	{
		texdata = tex2DLayered(texref,gidx,gidy,gidz);
		mydata = data(gidx,gidy,gidz);

		printf(" mydata = %f, texdata = %f @ %i, %i\n",mydata,texdata,gidx,gidy);

	}


}







int main(void)
{
	int nx = 4;
	int ny = 4;
	int nz = 2;

	int3 dims;
	dims.x = nx;
	dims.y = ny;
	dims.z = nz;


	dim3 cudaGridSize(1,1,2);
	dim3 cudaBlockSize(8,8,1);

	hipError_t status;

	cudaMatrixf initial_data(nx,ny,nz);
	cudaMatrixT<hiprandState> random_states(nx,ny,nz);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipExtent extent;
	extent.width = nx; // Note, for cudaArrays the width field is the width in elements, not bytes
	extent.height = ny;
	extent.depth = nz;

	hipArray *array = 0;



	status = hipMalloc3DArray(&array,&desc,extent,hipArrayLayered);

	if(status != hipSuccess){fprintf(stderr, " malloc array %s\n", hipGetErrorString(status));}

	printf("setup kernel \n");
	setup_kernel<<<cudaGridSize,cudaBlockSize>>>(random_states);
	hipDeviceSynchronize();
	status = hipGetLastError();
	if(status != hipSuccess){fprintf(stderr, " setup_kernel %s\n", hipGetErrorString(status));}

	printf("Fill kernel \n");
	fill_kernel<<<cudaGridSize,cudaBlockSize>>>(initial_data,dims,random_states);
	hipDeviceSynchronize();
	status = hipGetLastError();
	if(status != hipSuccess){fprintf(stderr, " fill kernel %s\n", hipGetErrorString(status));}

	hipPitchedPtr matrixPtr = initial_data.getptr();

	hipMemcpy3DParms params = {0};
	params.srcPtr = matrixPtr;
	params.dstArray = array;
	params.kind = hipMemcpyDeviceToDevice;
	params.extent = extent;

	status = hipMemcpy3D(&params);
	hipDeviceSynchronize();
	if(status != hipSuccess){fprintf(stderr, " copy array %s\n", hipGetErrorString(status));}



	hipBindTextureToArray(texref,array);
	hipDeviceSynchronize();
	status = hipGetLastError();
	if(status != hipSuccess){fprintf(stderr, " bind array %s\n", hipGetErrorString(status));}



	check_kernel<<<cudaGridSize,cudaBlockSize>>>(initial_data,dims);
	hipDeviceSynchronize();
	status = hipGetLastError();
	if(status != hipSuccess){fprintf(stderr, " check kernel %s\n", hipGetErrorString(status));}

	return 0;






}






































