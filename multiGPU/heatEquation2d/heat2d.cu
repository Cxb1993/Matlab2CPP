#include "hip/hip_runtime.h"

#include "heat2d.h"

void Manage_Memory(int phase, int tid, float **h_u, float **d_u, float **d_un){
  hipError_t Error;
  size_t global= ( NX+2)*sizeof(float);
  size_t local = (SNX+2)*sizeof(float);
  if (phase==0) {
    // Allocate domain on host
    *h_u = (float*)malloc(global);
   }
  if (phase==1) {
    // Allocate local domain variable on host
    //*h_ul = (float*)malloc(local);
    // Allocate local domain variable on device
    Error = hipSetDevice(tid);
    if (DEBUG) printf("CUDA error (hipSetDevice) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_u,local);
    if (DEBUG) printf("CUDA error (hipMalloc d_u) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_un,local);
    if (DEBUG) printf("CUDA error (hipMalloc d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
   }
  if (phase==2) {
    // Free local domain in on host
    //free(*h_ul);
    // Free local domain variable on device
    Error = hipFree(*d_u);
    if (DEBUG) printf("CUDA error (hipFree d_u) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipFree(*d_un);
    if (DEBUG) printf("CUDA error (hipFree d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
  }
  if (phase==3) {
    // Free the domain on host
    free(*h_u);
  }
}

void Manage_Comms(int phase, int tid, float **h_u, float **d_u){
  hipError_t Error;
  if (phase==1) {
    // Copy left and right cells from local domain to global domain
    // h_u[ 1 +tid*SNX] = t_u[ 1 ];
    // h_u[SNX+tid*SNX] = t_u[SNX];
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_u+1+tid*SNX,*d_u+1,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*h_u+SNX+tid*SNX,*d_u+SNX,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
  if (phase==2) {
    // Copy left and right cells from global domain to local domain
    // t_u[  0  ] = h_u[  0  +tid*SNX];
    // t_u[SNX+1] = h_u[SNX+1+tid*SNX];
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*d_u,*h_u+tid*SNX,sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*d_u+SNX+1,*h_u+SNX+1+tid*SNX,sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
  }
  if (phase==3) {
    // Transfer all data from local domains to global domain
    //for (int i = 0; i < SNX; i++) {
    //  h_u[i+1+tid*SNX] = t_u[i+1];
    //}
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_u+1+tid*SNX,*d_u+1,SNX*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
}


void Set_IC(float *u0){
  // Set initial condition in global domain
  // for (int i = 1; i < NX+1; i++) u0[i] = 0.33;
  // Set Dirichlet boundary conditions in global domain
  u0[0]=0.0;  u0[NX+1]=1.0;
}

void Call_Init(float **u0){
  // Load the initial condition
  Set_IC(*u0);
}

__global__ void Set_GPU_IC(int tid, float *ut0){
  // local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  // set initial condition only at "interior" nodes
  if (i>0 & i<SNX+1) {
    //switch (tid) { 
    //case 0: ut0[i] = 0.10; break;
    //case 1: ut0[i] = 0.25; break;
    //case 2: ut0[i] = 0.40; break;
    //case 3: ut0[i] = 0.50; break;
    //case 4: ut0[i] = 0.75; break;
    //case 5: ut0[i] = 0.90; break;    
    // }
   ut0[i] = 0.0;
  }
}

void Call_GPU_Init(int tid, float **ut0){
  // Load the initial condition
  int threads = 128;
  int blocks = ((SNX+2) + threads - 1)/threads;
  Set_GPU_IC<<<blocks,threads>>>(tid,*ut0);
  if (DEBUG) printf("CUDA error (Set_GPU_IC) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
}

__global__ void Laplace1d(float *u, float *un){
  // local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  int o =   i  ; // node( j,i ) 
  int r = (i+1); // node(j-1,i)  l--o--r
  int l = (i-1); // node(j,i-1) 

  // only update "interior" nodes
  if(i>0 && i<SNX+1) {
    un[o] = u[o] + KX*(u[r]-2*u[o]+u[l]);
  } else {
    un[o] = u[o];
  }
}

void Call_Laplace(int tid, float **u, float **un){
  // Produce one iteration of the laplace operator
  int threads = 128;
  int blocks = ((SNX+2) + threads - 1)/threads;
  Laplace1d<<<blocks,threads>>>(*u,*un);
  if (DEBUG) printf("CUDA error (Call_Laplace) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
}

void Update_Domain(int tid, float *h_u, float *t_u){
  // Explicitly copy data arrays
  if (DEBUG) printf("Copying thread data into the whole domain (thread %d)\n",tid); 
  for (int i = 0; i < SNX; i++) {
    h_u[i+1+tid*SNX] = t_u[i+1];
  }
}

void Call_Update(int tid, float **h_u, float **t_u){
  // produce explicitly: h_u = t_u
  Update_Domain(tid,*h_u,*t_u);
}

void Save_Results(float *u){
  // print result to txt file
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int i = 0; i < NX+2; i++) {
      fprintf(pFile, "%d\t %g\n",i,u[i]);
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}
