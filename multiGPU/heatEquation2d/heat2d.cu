#include "hip/hip_runtime.h"

#include "heat2d.h"

void Manage_Memory(int phase, int tid, float **h_u, float **h_ul, float **d_u, float **d_un){
  hipError_t Error;
  size_t global= ( NX+2)*( NY+2)*sizeof(float);
  size_t local = (SNX+2)*(SNY+2)*sizeof(float);
  if (phase==0) {
    // Allocate domain on host
    *h_u = (float*)malloc(global);
   }
  if (phase==1) {
    // Allocate local domain variable on device
    *h_ul = (float*)malloc(local);
    Error = hipSetDevice(tid); if (DEBUG) printf("CUDA error (hipSetDevice) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_u ,local); if (DEBUG) printf("CUDA error (hipMalloc d_u ) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_un,local); if (DEBUG) printf("CUDA error (hipMalloc d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
   }
  if (phase==2) {
    // Free local domain variable on device
    Error = hipFree(*d_u ); if (DEBUG) printf("CUDA error (hipFree d_u ) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipFree(*d_un); if (DEBUG) printf("CUDA error (hipFree d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
    free(*h_ul);
  }
  if (phase==3) {
    // Free the domain on host
    free(*h_u);
  }
}

void Manage_Comms(int phase, int tid, float **h_u, float **h_ul, float **d_u){
  hipError_t Error;
  if (phase==1) {
    // Copy left, right, up and down "interior" boundary  cells from local domain to global domain
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_u+ 1 +tid*SNX+(NX+2)* 1 ,*d_u+ 1 +(SNX+2)* 1 ,SNX*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*h_u+SNX+tid*SNX+(NX+2)*SNY,*d_u+SNX+(SNX+2)*SNY,SNX*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    for (int j = 1; j < SNY+1; j++) {
      Error=hipMemcpy(*h_u+ 1 +tid*SNX+(NX+2)*j,*d_u+ 1 +(SNX+2)*j,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
      Error=hipMemcpy(*h_u+SNX+tid*SNX+(NX+2)*j,*d_u+SNX+(SNX+2)*j,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    }
  }
  if (phase==2) {
    // Copy left, right, up and down boundary cells from global domain to local domain
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*d_u+  0  +(SNX+2)* 1 ,*h_u+  0  +tid*SNX+(NX+2)* 1 ,SNX*sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*d_u+SNX+1+(SNX+2)*SNY,*h_u+SNX+1+tid*SNX+(NX+2)*SNY,SNX*sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
    for (int j = 1; j < SNY+1; j++) {
      Error=hipMemcpy(*d_u+  0  +(SNX+2)*j,*h_u+  0  +tid*SNX+(NX+2)*j,sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
      Error=hipMemcpy(*d_u+SNX+1+(SNX+2)*j,*h_u+SNX+1+tid*SNX+(NX+2)*j,sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy h -> d) = %s \n",hipGetErrorString(Error));
    }
  }
  if (phase==3) {
    // Transfer all data from local domains to global domain
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    for (int j = 1; j < SNY+1; j++) {
      Error=hipMemcpy(*h_u+ 1 +tid*SNX+(NX+2)*j,*d_u+ 1 +(SNX+2)*j,SNX*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    }
  }
  if (phase==4) {
    // Transfer all data from local domains to global domain
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_ul,*d_u,(SNX+2)*(SNY+2)*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
}


void Set_IC(float *u0){
  // Set Dirichlet boundary conditions in global domain
  for (int i = 0; i < NX+2; i++) u0[   i  +(NX+2)*   0  ]=0.0; // down
  for (int i = 1; i < NX+2; i++) u0[   i  +(NX+2)*(NY+1)]=1.0; // up
  for (int j = 0; j < NY+2; j++) u0[   0  +(NX+2)*   j  ]=0.0; // left
  for (int j = 1; j < NY+2; j++) u0[(NX+1)+(NX+2)*   j  ]=1.0; // right
}

void Call_Init(float **u0){
  // Load the initial condition
  Set_IC(*u0);
}

__global__ void Set_GPU_IC(int tid, float *u){
  // Build local threads indexes
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int o = i+(SNX+2)*j; u[o] = 0.0;

  // Set initial condition only at "interior" nodes
  if (o<(SNX+2)*(SNY+2)) {
    if (i>0 && i<SNX+1 && j>0 && j<SNY+1) {
      if      (tid==0) u[o] = 0.10;
      else if (tid==1) u[o] = 0.25;
      else if (tid==2) u[o] = 0.40;
      else if (tid==3) u[o] = 0.50;
      else if (tid==4) u[o] = 0.75;
      else if (tid==5) u[o] = 0.90; 
    }
  }
}

void Call_GPU_Init(int tid, float **ut0){
  // Load the initial condition
  dim3 threads(16,16);
  dim3 blocks((SNX+2)/16,(SNY+2)/16);
  Set_GPU_IC<<<blocks,threads>>>(tid,*ut0);
  if (DEBUG) printf("CUDA error (Set_GPU_IC) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
  hipError_t Error = hipDeviceSynchronize();
  if (DEBUG) printf("CUDA error (Set_GPU_IC Synchronize) %s\n",hipGetErrorString(Error));
}

__global__ void Laplace1d(float *u, float *un){
  // local threads indexes
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int o = (i + (SNX+2)*j); // node( j,i )     n
  int n = i+(SNX+2)*(j+1); // node(j+1,i)     |
  int s = i+(SNX+2)*(j-1); // node(j-1,i)  w--o--e
  int e = (i+1)+(SNX+2)*j; // node(j,i+1)     |
  int w = (i-1)+(SNX+2)*j; // node(j,i-1)     s

  // only update "interior" nodes
  if(i>0 & i<SNX+1 & j>0 & j<SNY+1) {
    un[o] = u[o] + KX*(u[e]-2*u[o]+u[w]) + KY*(u[n]-2*u[o]+u[s]);
  } else {
    un[o] = u[o];
  }
}

void Call_Laplace(int tid, float **u, float **un){
  // Produce one iteration of the laplace operator
  dim3 threads(16,16);
  dim3 blocks((SNX+2)/16,(SNY+2)/16);
  Laplace1d<<<blocks,threads>>>(*u,*un);
  if (DEBUG) printf("CUDA error (Call_Laplace) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
}

void Save_Results_Tid(int tid, float *u){
  // print result to txt file
  if (tid==0) {
    FILE *pFile = fopen("result0.txt", "w");
    if (pFile != NULL) {
      for (int j = 0; j < SNY+2; j++) {
	for (int i = 0; i < SNX+2; i++) {
	  fprintf(pFile, "%d\t %d\t %g\n",i,j,u[i+(SNX+2)*j]);
	}
      }
      fclose(pFile);
    } else {
      printf("Unable to save to file\n");
    }
  }
  if (tid==1) {
    FILE *pFile = fopen("result1.txt", "w");
    if (pFile != NULL) {
      for (int j = 0; j < SNY+2; j++) {
	for (int i = 0; i < SNX+2; i++) {
	  fprintf(pFile, "%d\t %d\t %g\n",i,j,u[i+(SNX+2)*j]);
	}
      }
      fclose(pFile);
    } else {
      printf("Unable to save to file\n");
    }
  }
}

void Save_Results(float *u){
  // print result to txt file
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int j = 0; j < NY+2; j++) {
      for (int i = 0; i < NX+2; i++) {      
	fprintf(pFile, "%d\t %d\t %g\n",i,j,u[i+(NX+2)*j]);
      }
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}
