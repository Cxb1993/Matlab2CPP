#include "hip/hip_runtime.h"

#include "heat1d.h"

void Manage_Memory(int phase, int tid, float **h_u, float **h_ul, float **d_u, float **d_un){
  hipError_t Error;
  if (phase==0) {
    // Allocate domain on host
    *h_u = (float*)malloc((NX+2)*sizeof(float));
   }
  if (phase==1) {
    // Allocate local domain variable on host
    *h_ul = (float*)malloc((SNX+2)*sizeof(float));
    // Allocate local domain variable on device
    Error = hipSetDevice(tid);
    if (DEBUG) printf("CUDA error (hipSetDevice) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_u,(SNX+2)*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc d_u) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipMalloc((void**)d_un,(SNX+2)*sizeof(float));
    if (DEBUG) printf("CUDA error (hipMalloc d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
   }
  if (phase==2) {
    // Free local domain in on host
    free(*h_ul);
    // Free local domain variable on device
    Error = hipFree(*d_u);
    if (DEBUG) printf("CUDA error (hipFree d_u) in thread %d = %s\n",tid,hipGetErrorString(Error));
    Error = hipFree(*d_un);
    if (DEBUG) printf("CUDA Error (hipFree d_un) in thread %d = %s\n",tid,hipGetErrorString(Error));
  }
  if (phase==3) {
    // Free the domain on host
    free(*h_u);
  }
}

void Manage_Comms(int phase,int tid, float **h_u, float ***h_ul, float **d_u){
  hipError_t Error;
  if (phase==1) {
    // Copy left and right cells from local domain to global domain
    // h_u[ 1 +tid*SNX] = t_u[ 1 ];
    // h_u[SNX+tid*SNX] = t_u[SNX];
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_u+1+tid*SNX,*d_u+1,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*h_u+SNX+tid*SNX,*d_u+SNX,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
  if (phase==2) {
    // Copy left and right cells from global domain to local domain
    // t_u[  0  ] = h_u[  0  +tid*SNX];
    // t_u[SNX+1] = h_u[SNX+1+tid*SNX];
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*d_u,*h_u+tid*SNX,sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
    Error=hipMemcpy(*d_u+SNX+1,*h_u+SNX+1+tid*SNX,sizeof(float),hipMemcpyHostToDevice); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
  if (phase==3) {
    // Transfer all data from local domains to global domain
    //for (int i = 0; i < SNX; i++) {
    //  h_u[i+1+tid*SNX] = t_u[i+1];
    //}
    if (DEBUG) printf("::: Perform GPU-CPU comms (phase %d, thread %) :::\n",phase,tid);
    Error=hipMemcpy(*h_u+1+tid*SNX,*d_u+1,SNX*sizeof(float),hipMemcpyDeviceToHost); if (DEBUG) printf("CUDA error (Memcpy d -> h) = %s \n",hipGetErrorString(Error));
  }
}


void Set_IC(float *u0){
  // Set initial condition in global domain
  for (int i = 1; i < NX+1; i++) {u0[i] = 0.0;}  u0[0]=0.0;  u0[NX+1]=1.0;
}

void Call_Init(float **u0){
  // Load the initial condition
  Set_IC(*u0);
}

__global__ void Set_GPU_IC(int tid,float *ut0){
  // Set domain initial condition in local threads
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  ut0[i] = 0.0;

}

void Call_GPU_Init(int tid,float **ut0){
  // Load the initial condition
  int threads = 64;
  int blocks = (N_GPU + threads - 1)/threads;
  Set_GPU_IC<<<blocks,threads>>>(tid,*ut0);
  if (DEBUG) printf("CUDA error (Set_GPU_IC) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
}

__global__ void Laplace1d(float *u,float *un){
  // Using (i,j) = [i+N*j] indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  int o =   i  ; // node( j,i ) 
  int r = (i+1); // node(j-1,i)  l--o--r
  int l = (i-1); // node(j,i-1) 

  // only update "interior" nodes
  if(i>0 && i<SNX+1) {
    un[o] = u[o] + KX*(u[r]-2*u[o]+u[l]);
  } else {
    un[o] = u[o];
  }
}

void Call_Laplace(int tid, float **u, float **un){
  // Produce one iteration of the laplace operator
  int threads = 64;
  int blocks = (N_GPU + threads - 1)/threads;
  Laplace1d<<<blocks,threads>>>(*u,*un);
  if (DEBUG) printf("CUDA error (Set_GPU_IC) in thread %d = %s\n",tid,hipGetErrorString(hipPeekAtLastError()));
}

void Update_Domain(int tid, float *h_u, float *t_u){
  // Explicitly copy data arrays
  if (DEBUG) printf("Copying thread data into the whole domain (thread %d)\n",tid); 
  for (int i = 0; i < SNX; i++) {
    h_u[i+1+tid*SNX] = t_u[i+1];
  }
}

void Call_Update(int tid, float **h_u, float **t_u){
  // produce explicitly: h_u = t_u
  Update_Domain(tid,*h_u,*t_u);
}

void Save_Results(float *u){
  // print result to txt file
  FILE *pFile = fopen("result.txt", "w");
  if (pFile != NULL) {
    for (int i = 0; i < NX+2; i++) {
      fprintf(pFile, "%d\t %g\n",i,u[i]);
    }
    fclose(pFile);
  } else {
    printf("Unable to save to file\n");
  }
}
